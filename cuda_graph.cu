
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include <queue>
#include <vector>
#include <functional>
#include <cfloat>

#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/op_kernel.h"
#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/shape_inference.h"
#include "tensorflow/core/framework/common_shape_fns.h"

using namespace tensorflow;

typedef Eigen::ThreadPoolDevice CPUDevice;
typedef Eigen::GpuDevice GPUDevice;

typedef unsigned int uint32;
typedef unsigned long long int uint64;
const int __INT_MAX = 0x7fffffff;

////////////////////////////////////////////////////////////////////////////////
// This class is defined for the convenience of priority_queue
// The attribute x and y are for saving the indices of two nodes,
// and the attribute dist is for saving and comparing the distance between the nodes.
template <class T> class Node_t {
public:
	T dist;
	uint64 x, y;

	Node_t() {}
	Node_t(T _d, uint64 _x, uint64 _y) : dist(_d), x(_x), y(_y) {}
	__device__ bool operator<(const Node_t<T>& n) const { return dist < n.dist; }
	__device__ bool operator>(const Node_t<T>& n) const { return dist > n.dist; }
	__device__ bool operator<=(const Node_t<T>& n) const { return dist <= n.dist; }
	__device__ bool operator>=(const Node_t<T>& n) const { return dist >= n.dist; }
};

/////////////////////////////////////////////////////////////////////////////
// Here we begin our implementation of the graph adjacency matrix generation.
template <typename T>
__device__ __inline__ T distance(const T* a, const T* b) {
	return (a[0]-b[0])*(a[0]-b[0]) + (a[1]-b[1])*(a[1]-b[1]) + (a[2]-b[2])*(a[2]-b[2]);
}

// The CUDA kernel of the implementation
template <typename T>
__global__ void GraphGenerator(const T* in, int* out, int k_val, uint64 num_points, uint64 channels) {
	// Since the input has shape [batch_size, num_points, channels], 
	// we use batch_size numbers of blocks, and num_points numbers of threads.
	uint64 in_begin = blockIdx.x * blockDim.x * channels;
	uint64 out_begin = blockIdx.x * blockDim.x * k_val + threadIdx.x * k_val;
	T* heap = new T[k_val];	

	// Initialization of the output tensor
	for(int i=0; i<k_val; i++) {
		out[out_begin + i] = __INT_MAX;
	}

	// The time complexity is not optimal, yet k_val is often a small integer,
	// this solution may also be acceptable.
	for(int i=0; i<num_points; i++) {
		if(i == threadIdx.x)
			continue;
		T tmp = distance(&in[in_begin + threadIdx.x*channels], &in[in_begin + i*channels]);
		for(int j=0; j<k_val; j++) {
			if(tmp < heap[j]) {
				heap[j] = tmp;
				out[out_begin + j] = i;
			}
		}
	}
	delete heap;
}

template <typename T>
void GraphLauncher(const T* input, int* output, int k_val, uint64 batch_size, uint64 num_points, uint64 channels) {
	GraphGenerator<<< batch_size, num_points >>>(input, output, k_val, num_points, channels);
	cudaDeviceSynchronize();
}


////////////////////////////////////////////////////////////////////////////////////
// The tensorflow OpKernel implementation wrapper.
template <typename Device, typename T>
class GraphAdjacencyGeneratorOp : public OpKernel {
public:
	explicit GraphAdjacencyGeneratorOp(OpKernelConstruction* context) : OpKernel(context) {
		OP_REQUIRES_OK(context, context->GetAttr("K", &K_));
		//OP_REQUIRES(context, (K_>0 && K_<=num_points), 
		//	::tensorflow::errors::InvalidArgument("The value of K must be an integer in range [1, num_points]"));
	}

	void Compute(OpKernelContext* context) override {
		const Tensor& input_tensor = context->input(0);
		OP_REQUIRES(context, input_tensor.dims() == 3,
			::tensorflow::errors::InvalidArgument("GraphAdjacencyGenerator expects (batch_size, num_points, 3) points shape"));
		uint64 batch_size = input_tensor.shape().dim_size(0);
		uint64 num_points = input_tensor.shape().dim_size(1);
		uint64 channels = input_tensor.shape().dim_size(2);
		OP_REQUIRES(context, channels == 3,
			::tensorflow::errors::InvalidArgument("The last dimension for the input op must be 3"));
		auto input = input_tensor.flat<T>();

		// Validate the value of K_
		//OP_REQUIRES_OK(context, context->GetAttr("K"), &K_);
		OP_REQUIRES(context, (K_>0 && K_<=num_points), 
			::tensorflow::errors::InvalidArgument("The value of K must be an integer in range [1, num_points]"));

		Tensor* output_tensor = NULL;
		::tensorflow::TensorShape output_shape;
		output_shape.AddDim(batch_size);
		output_shape.AddDim(num_points);
		output_shape.AddDim(K_);
		OP_REQUIRES_OK(context, context->allocate_output(0, output_shape, &output_tensor));
		auto output = output_tensor->template flat<int>();

		// GraphLaucher(input.data(), output.data(), k_val, num_points, channels);
		GraphLauncher<T>(input.data(), output.data(), K_, batch_size, num_points, channels);
	}

private:
	int K_;

};

REGISTER_KERNEL_BUILDER(Name("GraphAdjacencyGenerator").Device(DEVICE_GPU).TypeConstraint<float>("T"), GraphAdjacencyGeneratorOp<GPUDevice, float>);
//REGISTER_KERNEL_BUILDER(Name("GraphAdjacencyGenerator").Device(DEVICE_GPU).TypeConstraint<double>("T"), GraphAdjacencyGeneratorOp<GPUDevice, double>);

#endif
